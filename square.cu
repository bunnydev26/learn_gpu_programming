
#include <hip/hip_runtime.h>
#include <stdio.h>

// nvcc -o square square.cu

__global__ void square(float * d_out, float * d_in) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    // printf("%f --\n",f);
    d_out[idx] = f * f;
}

int main(int argc, char ** argv) {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    for(int i=0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    //declare gpu memory pointers
    float * d_in;
    float * d_out;

    //allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // transfering the array to gpu
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // Launch the kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // Copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // Print out the result array
    for(int i=0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i%4) != 3) ? "\t" : "\n");
    }

    // free GPU allocation
    hipFree(d_in);
    hipFree(d_out);

    return 0;

}